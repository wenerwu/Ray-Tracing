#include "hip/hip_runtime.h"
#include "cudacudaMatrix3x3.h"

#include <iostream>
#include <cmath>

using namespace std;

namespace CMU462 {

  // double& cudaMatrix3x3::operator()( int i, int j ) {
  //   return entries[j][i];
  // }

  // const double& cudaMatrix3x3::operator()( int i, int j ) const {
  //   return entries[j][i];
  // }

  // cudaVector3D& cudaMatrix3x3::operator[]( int j ) {
  //     return entries[j];
  // }

  // const cudaVector3D& cudaMatrix3x3::operator[]( int j ) const {
  //   return entries[j];
  // }

  void cudaMatrix3x3::zero( double val ) {
    // sets all elements to val
    entries[0] = entries[1] = entries[2] = cudaVector3D( val, val, val );
  }

  double cudaMatrix3x3::det( void ) const {
    const cudaMatrix3x3& A( *this );

    return -A(0,2)*A(1,1)*A(2,0) + A(0,1)*A(1,2)*A(2,0) +
            A(0,2)*A(1,0)*A(2,1) - A(0,0)*A(1,2)*A(2,1) -
            A(0,1)*A(1,0)*A(2,2) + A(0,0)*A(1,1)*A(2,2) ;
  }

  double cudaMatrix3x3::norm( void ) const {
    return sqrt( entries[0].norm2() +
                 entries[1].norm2() +
                 entries[2].norm2() );
  }

  cudaMatrix3x3 cudaMatrix3x3::operator-( void ) const {

   // returns -A
    const cudaMatrix3x3& A( *this );
    cudaMatrix3x3 B;

    B(0,0) = -A(0,0); B(0,1) = -A(0,1); B(0,2) = -A(0,2);
    B(1,0) = -A(1,0); B(1,1) = -A(1,1); B(1,2) = -A(1,2);
    B(2,0) = -A(2,0); B(2,1) = -A(2,1); B(2,2) = -A(2,2);

    return B;
  }

  void cudaMatrix3x3::operator+=( const cudaMatrix3x3& B ) {

    cudaMatrix3x3& A( *this );
    double* Aij = (double*) &A;
    const double* Bij = (const double*) &B;

    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
    *Aij++ += *Bij++;
  }

  cudaMatrix3x3 cudaMatrix3x3::operator-( const cudaMatrix3x3& B ) const {
    const cudaMatrix3x3& A( *this );
    cudaMatrix3x3 C;

    for( int i = 0; i < 3; i++ )
    for( int j = 0; j < 3; j++ )
    {
       C(i,j) = A(i,j) - B(i,j);
    }

    return C;
  }

  cudaMatrix3x3 cudaMatrix3x3::operator*( double c ) const {
    const cudaMatrix3x3& A( *this );
    cudaMatrix3x3 B;

    for( int i = 0; i < 3; i++ )
    for( int j = 0; j < 3; j++ )
    {
       B(i,j) = c*A(i,j);
    }

    return B;
  }

  cudaMatrix3x3 operator*( double c, const cudaMatrix3x3& A ) {

    cudaMatrix3x3 cA;
    const double* Aij = (const double*) &A;
    double* cAij = (double*) &cA;

    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);
    *cAij++ = c * (*Aij++);

    return cA;
  }

  cudaMatrix3x3 cudaMatrix3x3::operator*( const cudaMatrix3x3& B ) const {
    const cudaMatrix3x3& A( *this );
    cudaMatrix3x3 C;

    for( int i = 0; i < 3; i++ )
    for( int j = 0; j < 3; j++ )
    {
       C(i,j) = 0.;

       for( int k = 0; k < 3; k++ )
       {
          C(i,j) += A(i,k)*B(k,j);
       }
    }

    return C;
  }

  // __device__ cudaVector3D cudaMatrix3x3::operator*( const cudaVector3D& x ) const {
  //   return x[0]*entries[0] +
  //          x[1]*entries[1] +
  //          x[2]*entries[2] ;
  // }
  
  // __device__ cudaMatrix3x3 cudaMatrix3x3::T( void ) const {
  //   const cudaMatrix3x3& A( *this );
  //   cudaMatrix3x3 B;

  //   for( int i = 0; i < 3; i++ )
  //   for( int j = 0; j < 3; j++ )
  //   {
  //      B(i,j) = A(j,i);
  //   }

  //   return B;
  // }

  cudaMatrix3x3 cudaMatrix3x3::inv( void ) const {
    const cudaMatrix3x3& A( *this );
    cudaMatrix3x3 B;

    B(0,0) = -A(1,2)*A(2,1) + A(1,1)*A(2,2); B(0,1) =  A(0,2)*A(2,1) - A(0,1)*A(2,2); B(0,2) = -A(0,2)*A(1,1) + A(0,1)*A(1,2);
    B(1,0) =  A(1,2)*A(2,0) - A(1,0)*A(2,2); B(1,1) = -A(0,2)*A(2,0) + A(0,0)*A(2,2); B(1,2) =  A(0,2)*A(1,0) - A(0,0)*A(1,2);
    B(2,0) = -A(1,1)*A(2,0) + A(1,0)*A(2,1); B(2,1) =  A(0,1)*A(2,0) - A(0,0)*A(2,1); B(2,2) = -A(0,1)*A(1,0) + A(0,0)*A(1,1);

    B /= det();

    return B;
  }

  void cudaMatrix3x3::operator/=( double x ) {
    cudaMatrix3x3& A( *this );
    double rx = 1./x;

    for( int i = 0; i < 3; i++ )
    for( int j = 0; j < 3; j++ )
    {
       A( i, j ) *= rx;
    }
  }

  cudaMatrix3x3 cudaMatrix3x3::identity( void ) {
    cudaMatrix3x3 B;

    B(0,0) = 1.; B(0,1) = 0.; B(0,2) = 0.;
    B(1,0) = 0.; B(1,1) = 1.; B(1,2) = 0.;
    B(2,0) = 0.; B(2,1) = 0.; B(2,2) = 1.;

    return B;
  }

  cudaMatrix3x3 cudaMatrix3x3::crossProduct( const cudaVector3D& u ) {
    cudaMatrix3x3 B;

    B(0,0) =   0.;  B(0,1) = -u.z;  B(0,2) =  u.y;
    B(1,0) =  u.z;  B(1,1) =   0.;  B(1,2) = -u.x;
    B(2,0) = -u.y;  B(2,1) =  u.x;  B(2,2) =   0.;

    return B;
  }

  cudaMatrix3x3 outer( const cudaVector3D& u, const cudaVector3D& v ) {
    cudaMatrix3x3 B;
    double* Bij = (double*) &B;

    *Bij++ = u.x*v.x;
    *Bij++ = u.y*v.x;
    *Bij++ = u.z*v.x;
    *Bij++ = u.x*v.y;
    *Bij++ = u.y*v.y;
    *Bij++ = u.z*v.y;
    *Bij++ = u.x*v.z;
    *Bij++ = u.y*v.z;
    *Bij++ = u.z*v.z;

    return B;
  }

  std::ostream& operator<<( std::ostream& os, const cudaMatrix3x3& A ) {
    for( int i = 0; i < 3; i++ )
    {
       os << "[ ";

       for( int j = 0; j < 3; j++ )
       {
          os << A(i,j) << " ";
       }

       os << "]" << std::endl;
    }

    return os;
  }

  cudaVector3D& cudaMatrix3x3::column( int i ) {
    return entries[i];
  }

  const cudaVector3D& cudaMatrix3x3::column( int i ) const {
    return entries[i];
  }
}
