#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

#include "CMU462/CMU462.h"
#include "CMU462/vector3D.h"
#include "CMU462/matrix3x3.h"
#include "CMU462/lodepng.h"

#include "../static_scene/sphere.h"
#include "../static_scene/triangle.h"
#include "../static_scene/light.h" 
#include "../static_scene/scene.h"

#include "cudaPathtracer.h"


#include "cudaSpectrum.h"
#include "cudabsdf.h"
#include "cudaintersection.h"
#include "cudaPrimitive.h" 
#include "cudaTriangle.h"
#include "cudaCamera.h"
#include "cudaMatrix3x3.h"
#include "cudaRay.h"
#include "cudaLight.h"
#include "cudaBVH.h"


//using namespace std;
using namespace CMU462;
using namespace StaticScene;


using std::min;
using std::max;


#define BLOCKSIZE 256
#define MAXLIGHT 16
#define MAXSTACK 256


__constant__ int primitiveCount;
cudaTriangle* primitives;
__constant__ int lightCount;
cudaComplexLight* cudaLights;
__constant__ double sensorHeight; 
__constant__ double sensorWidth; 
__constant__ size_t width;
__constant__ size_t height;
__constant__ double INFD;

PathTracer* pathtracer;
cudaSpectrum* spectrum_buffer;
cudaPrimitive* cudaPrimitives;
cudaCamera* camera;  
cudaMatrix3x3 c2w;
cudaBVHNode* root;



cudaPathTracer::cudaPathTracer(PathTracer* _pathTracer) {
    pathtracer = _pathTracer;
 
}

cudaPathTracer::~cudaPathTracer() {
    hipFree(spectrum_buffer);
    hipFree(camera);
    // delete bvh;
    // delete gridSampler; 
    // delete hemisphereSampler;
}

cudaComplexLight translateLight(SceneLight* light)
{
    cudaComplexLight res = cudaComplexLight();
    res.type = light->get_type();
    // namespace cudaLightType {
    //   enum TYPE{ NONE, DIRECTIONAL, INFINITEHEMISPHERE, POINT, SPOT, AREA, SPHERE, MESH };
    // }

    DirectionalLight* dl;
    InfiniteHemisphereLight* il;
 
    switch(res.type)
    {
      case cudaLightType::DIRECTIONAL:
         dl = (DirectionalLight*)light;
        res.radiance = dl->radiance;
        res.dirToLight = dl->dirToLight;
        break;

      case cudaLightType::INFINITEHEMISPHERE:
         il = (InfiniteHemisphereLight*)light;
        res.radiance = il->radiance;
        res.sampleToWorld = il->sampleToWorld;
        res._3Dsampler = il->sampler;
        break;

    }

  //  printf("!!!!!!TYPE:%d\n", res.type);
  //  res.radiance = light->radiance;


    return res;
}

cudaBVHNode* loadBVHNode(BVHNode* node)
{
  if(!node)
    return NULL;

  cudaBVHNode cpuNode;
  cpuNode.bb = node->bb;   // todo CUDABBOX
  cpuNode.start = node->start;
  cpuNode.range = node->range;

  cpuNode.l = loadBVHNode(node->l);
  cpuNode.r = loadBVHNode(node->r);

  cudaBVHNode* newNode;
  hipMalloc(&newNode, sizeof(cudaBVHNode));
  hipMemcpy(newNode, &cpuNode, sizeof(cudaBVHNode), hipMemcpyHostToDevice);
  
  return newNode;
}

void loadBVH()
{
  hipError_t err;

  root = loadBVHNode(pathtracer->bvh->root);
 // hipMemcpyToSymbol(HIP_SYMBOL(root), tmpRoot,  sizeof(cudaBVHNode));

  err = hipPeekAtLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to init load bvh (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

void loadLights()
{
  hipError_t err;

  int lcount = pathtracer->scene->lights.size();
  hipMemcpyToSymbol(HIP_SYMBOL(lightCount), &lcount,  sizeof(int));

  err = hipPeekAtLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to init light count (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  hipMalloc(&cudaLights, sizeof(cudaComplexLight) * lcount);
  err = hipPeekAtLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to malloc light (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  cudaComplexLight* cpuLight = (cudaComplexLight *)malloc(sizeof(cudaComplexLight) * lcount);
  for(int i = 0; i < lcount; i++)
  {
    cpuLight[i] = translateLight(pathtracer->scene->lights[i]);
  //  printf("LIGHT TYPE:%d\n", cpuLight[i].type);
  }

  hipMemcpy(cudaLights, cpuLight, sizeof(cudaComplexLight) * lcount, hipMemcpyHostToDevice);
  //  hipMemcpyToSymbol(HIP_SYMBOL(primitives), cpuTriangle,  sizeof(cudaTriangle)  * prim_num);
    free(cpuLight);
  
    err = hipPeekAtLastError();
  
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to init light (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}

void loadPrimitives()
{
  hipError_t err;
  int prim_num = pathtracer->bvh->primitives.size(); 
  hipMemcpyToSymbol(HIP_SYMBOL(primitiveCount), &prim_num,  sizeof(int));
  err = hipPeekAtLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to init primitive count (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }


  hipMalloc(&primitives, sizeof(cudaTriangle) * prim_num);
  err = hipPeekAtLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to malloc primitive (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  cudaTriangle* cpuTriangle = (cudaTriangle *)malloc(prim_num * sizeof(cudaTriangle));
  // TODO: NEED TO REALLY TRANSLATE IT
  for(int i = 0; i < prim_num; i++)
  {
    Triangle* prim = (Triangle*)pathtracer->bvh->primitives[i];
    cpuTriangle[i].mesh = prim->mesh;
    cpuTriangle[i].v1 = prim->v1;
    cpuTriangle[i].v2 = prim->v2;
    cpuTriangle[i].v3 = prim->v3;
    cpuTriangle[i].v = prim->v;

    
    cpuTriangle[i].p0 = prim->mesh->positions[prim->v1]; 
    cpuTriangle[i].p1 = prim->mesh->positions[prim->v2];
    cpuTriangle[i].p2 = prim->mesh->positions[prim->v3];

    cpuTriangle[i].n0 = prim->mesh->normals[prim->v1]; 
    cpuTriangle[i].n1 = prim->mesh->normals[prim->v2]; 
    cpuTriangle[i].n2 = prim->mesh->normals[prim->v3]; 

    DiffuseBSDF* bsdf = (DiffuseBSDF*)prim->mesh->bsdf;
    cpuTriangle[i].bsdf.albedo = bsdf->albedo;
    cpuTriangle[i].bsdf.sampler = bsdf->sampler;

  //  printf("%f \n",cpuTriangle[i].mesh->positions[cpuTriangle[i].v1].x);
  }
  hipMemcpy(primitives, cpuTriangle, sizeof(cudaTriangle)  * prim_num, hipMemcpyHostToDevice);
//  hipMemcpyToSymbol(HIP_SYMBOL(primitives), cpuTriangle,  sizeof(cudaTriangle)  * prim_num);
  free(cpuTriangle);

  err = hipPeekAtLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to init primitive (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

void cudaPathTracer::set_scene(Scene *scene) {
  hipError_t err;
  double sh = 2 * tan(radians(pathtracer->camera->vFov) / 2) * 1;	// distance is always 1
  double sw = sh * pathtracer->camera->ar;


  hipMemcpyToSymbol(HIP_SYMBOL(sensorHeight), &sh,  sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(sensorWidth), &sw,  sizeof(double));


  int num = pathtracer->sampleBuffer.w * pathtracer->sampleBuffer.h;;
  //  spectrum_buffer = (Spectrum*)malloc(sizeof(Spectrum) * num);

    hipMalloc(&spectrum_buffer, sizeof(cudaSpectrum) * num);


    hipMalloc(&camera, sizeof(cudaCamera));
    hipMemcpy(camera, pathtracer->camera, sizeof(cudaCamera), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(width), &pathtracer->sampleBuffer.w,  sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(height), &pathtracer->sampleBuffer.h,  sizeof(double));
    
    err = hipPeekAtLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to init scene (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    loadPrimitives();
    loadLights();
    loadBVH();

    double infine =  std::numeric_limits<double>::infinity();
    hipMemcpyToSymbol(HIP_SYMBOL(INFD), &infine,  sizeof(double));
 // hipMalloc(&cudaPrimitives, sizeof(cudaPrimitive) * prim_num);
}

void cudaPathTracer::set_camera(Camera *camera) {
  //  pathtracer->set_camera(camera);
    // if (state != INIT) {
    // return;
    // }

    // this->camera = camera;
    // if (has_valid_configuration()) {
    // state = READY;
    // }
}

void cudaPathTracer::set_frame_size(size_t width, size_t height) {
    pathtracer->set_frame_size(width, height); 

}


void cudaPathTracer::update_screen() {
    pathtracer->update_screen(); 

  }

__device__ cudaVector3D getSample3D(hiprandState *state, size_t index)
{
  //TODO!!!!!!!!!!!!!!!!TRY FIND SOME SUBSTITUTE

  
  hiprandState local_state = state[index];
  double Xi1 = hiprand_uniform(&local_state);
  double Xi2 = hiprand_uniform(&local_state);
  state[index] = local_state;
  double theta = acos(Xi1);
  double phi = 2.0 * PI * Xi2;

  double xs = sinf(theta) * cosf(phi);
  double ys = sinf(theta) * sinf(phi);
  double zs = cosf(theta);

  return cudaVector3D(xs, ys, zs);
//  printf("index:%d %g %g\n", index, Xi1, Xi2);
//  return cudaVector3D(0.5, 0.5, 0.5);


}
__device__ cudaSpectrum sampleLight(cudaComplexLight* light, const cudaVector3D& p, cudaVector3D* wi, float* distToLight,
  float* pdf, hiprandState *state, size_t index)
{
  cudaSpectrum res;
  cudaVector3D dir;
  switch(light->type)
  {
    case cudaLightType::DIRECTIONAL:
      *wi = light->dirToLight;
      *distToLight = INFD;
      *pdf = 1.0;
      res = light->radiance;
    break;

    case cudaLightType::INFINITEHEMISPHERE:
      dir = getSample3D(state, index);
      *wi = light->sampleToWorld * dir;
      *distToLight = INFD;
      *pdf = 1.0 / (2.0 * M_PI);
      res = light->radiance;
    break;

    default:
    res = cudaSpectrum();
    break;
  }
  return res;
}


__device__ bool cudaintersectPrimitive(cudaTriangle* primitive, const cudaRay &r, cudaIntersection *isect)
{

    cudaVector3D p0 = primitive->p0; 
    cudaVector3D p1 = primitive->p1;
    cudaVector3D p2 = primitive->p2;
  
  
    cudaVector3D o = r.o;
    cudaVector3D d = r.d;
  
    cudaVector3D e1 = p1 - p0;
    cudaVector3D e2 = p2 - p0;
    cudaVector3D s = o - p0;
  
    double denominator = dot(cross(e1, d), e2);
    if (denominator == 0)
      return false;
  
    cudaVector3D numerator = cudaVector3D(-dot(cross(s, e2), d), dot(cross(e1, d), s), -dot(cross(s, e2), e1));
    cudaVector3D ans = numerator / denominator;
  //	return true;
    // in triangle
    if (ans.x < 0 || ans.x > 1 || ans.y < 0 || ans.y > 1 ||
      1 - ans.x - ans.y < 0 || 1 - ans.x - ans.y > 1 ||
      ans.z < r.min_t || ans.z > r.max_t)
      return false;
  
    double u = ans.x;
    double v = ans.y;
    double t = ans.z;
   
  //  cudaVector3D tt = cudaVector3D(t.x, t.y, t.z); 
    r.max_t = t;
  
    isect->t = t;

    cudaVector3D n0 = primitive->n0;
    cudaVector3D n1 = primitive->n1;
    cudaVector3D n2 = primitive->n2;
 
    cudaVector3D tmp = (1 - u - v) * n0 + u * n1 + v * n2;
     isect->n = cudaVector3D(tmp.x, tmp.y, tmp.z);
     if (dot(isect->n, r.d) > 0)
      isect->n *= -1;
    isect->primitive = primitive;
    isect->bsdf = &primitive->bsdf;	 
     
  return true; 
}

__device__ bool cudaintersectWithNode(const cudaRay &ray, cudaIntersection *isect, cudaTriangle* primitives, cudaBVHNode* root)
{
//	BVHNode* node = pathtracer->bvh->root;
  bool hit = false;

//  for (size_t p = 0; p < node->range; ++p) {
  for (size_t p = 0; p < primitiveCount; ++p) {
    if (cudaintersectPrimitive(&primitives[p], ray, isect))
//	if (pathtracer->bvh->primitives[node->start + p]->intersect(ray, isect))
  {
    hit = true;
  }
}
//   cudaBVHNode* s[MAXSTACK];
//   int stackSize = 0;

//   // TODO: FIRST NODE!
//   //stack<BVHNode*> s;
// 	double lt0, lt1, rt0, rt1;

// 	// TODO!!!
// //	int threadCount = 10;
// 	int pid = 0;
// 	int M[10];

// 	cudaBVHNode* near;
// 	cudaBVHNode* far;
  
//   cudaBVHNode* node = root;
// 	while(true)
// 	{
// 		// when it's leaf, intersect directly

// 		if(node->isLeaf())
// 		{	

// 			for (size_t p = 0; p < node->range; ++p) {
//         if (cudaintersectPrimitive(&primitives[node->start + p], ray, isect))
// 			//	if (pathtracer->bvh->primitives[node->start + p]->intersect(ray, isect))
// 				{
// 					hit = true;
// 				}
// 			}
// 			if(stackSize == 0)
// 				break;
// 			node = s[--stackSize];
//     }
    
// 		else
// 		{
// 			/* Parallel read ?*/
// 			int hitleft = (bool)node->l->bb.intersect(ray, lt0, lt1);
// 			int hitright = (bool)node->r->bb.intersect(ray, rt0, rt1);

// 			/* Use parallel and barrier to init */
// 			for(int i = 0; i <= 3; i++)
// 				M[i] = 0;

// 			// TODO: barrier here
// 			M[2*hitleft + hitright] = 1;
// 			// TODO: barrier here

// 			/* Visit both children */
// 			if(M[3] || (M[1] && M[2]))
// 			{
// 		//		printf("HERE!!\n");
// 				/* Decide which to go in first */
// 				M[pid] = 2 * (hitright && (rt0 < lt0)) - 1;

// 				/* TODO: PARLLEL SUM OVER HERE */
// 				if(M[pid] < 0)
// 				{
// 					near = node->l;
// 					far = node->r;
// 				}
// 				else
// 				{
// 					near = node->r;
// 					far = node->l;
//         }
//         s[stackSize++] = far;
// 				node = near;

// 			}
// 			else if(M[2])
// 			{
// 			//	printf("HERELEFT\n");
// 				node = node->l;
// 			}

// 			else if(M[1])
// 			{
// 			//	printf("HERERIGHT\n");
// 				node = node->r;
// 			}

// 			else
// 			{
// 				if(stackSize == 0)
// 					break;

//         node = s[--stackSize];
// 			}


// 		} // end else

// 	}

	return hit; 

}





__device__ cudaSpectrum trace_ray( const cudaRay &r, cudaTriangle* primitives, cudaComplexLight* lights, 
  hiprandState *state, size_t index, cudaBVHNode* root) 
  {
    cudaIntersection isect;  
   
   // if (!pathtracer->bvh->intersect(r, &isect)) {
    if (!cudaintersectWithNode(r, &isect, primitives, root)) {

        return cudaSpectrum(0, 0, 0);
    }

 //     return cudaSpectrum(1, 1, 1);
   // cudaSpectrum L_out =cudaSpectrum(1, 1, 1);
  //  cudaSpectrum L_out = (DiffuseBSDF)isect.bsdf->get_emission();  // Le
  cudaSpectrum L_out = cudaSpectrum();

    // TODO (PathTracer):
    // Instead of initializing this value to a constant color, use the direct,
    // indirect lighting components calculated in the code below. The starter
    // code overwrites L_out by (.5,.5,.5) so that you can test your geometry
    // queries before you implement path tracing.
  
    //L_out = Spectrum(5.f, 5.f, 5.f);
    //DirectionalLight dl = DirectionalLight(5, 100);
    
  
    cudaVector3D hit_p = r.o + r.d * isect.t;
    cudaVector3D hit_n = isect.n;
  
    // make a coordinate system for a hit point
    // with N aligned with the Z direction.
    cudaMatrix3x3 o2w;
    make_coord_space(o2w, isect.n);
    cudaMatrix3x3 w2o = o2w.T();
  
    // w_out points towards the source of the ray (e.g.,
    // toward the camera if this is a primary ray)
    cudaVector3D w_out = w2o * (r.o - hit_p);
    w_out.normalize();
  
  
  //  if (!isect.bsdf->is_delta()) {
      cudaVector3D dir_to_light;
      float dist_to_light;
      float pr;
  
      // ### Estimate direct lighting integral
      
    //  for (SceneLight* light : pathtracer->scene->lights) {
      for (int i = 0; i < lightCount; i++) {
        // no need to take multiple samples from a point/directional source
      //  int num_light_samples = light->is_delta_light() ? 1 : pathtracer->ns_area_light;
        cudaComplexLight* light = &lights[i];
        int num_light_samples = 1;
        // integrate light over the hemisphere about the normal
        for (int i = 0; i < num_light_samples; i++) {
  
          // returns a vector 'dir_to_light' that is a direction from
          // point hit_p to the point on the light source.  It also returns
          // the distance from point x to this point on the light source.
          // (pr is the probability of randomly selecting the random
          // sample point on the light source -- more on this in part 2)


          //  const cudaSpectrum& light_L = light->sample_L(hit_p, &dir_to_light, &dist_to_light, &pr);
          const cudaSpectrum& light_L = sampleLight(light, hit_p, &dir_to_light, &dist_to_light, &pr, state, index);

          // convert direction into coordinate space of the surface, where
          // the surface normal is [0 0 1]
          const cudaVector3D& w_in = w2o * dir_to_light;
          if (w_in.z < 0) continue;
  
            // note that computing dot(n,w_in) is simple
          // in surface coordinates since the normal is (0,0,1)
          double cos_theta = w_in.z;
            
          // evaluate surface bsdf
         // const cudaSpectrum& f = ((cudaDiffuseBSDF*)isect.bsdf)->f(w_out, w_in);
            const cudaSpectrum& f =  (isect.bsdf)->albedo * (1.0 / PI);
          // TODO (PathTracer):
          // (Task 4) Construct a shadow ray and compute whether the intersected surface is
          // in shadow. Only accumulate light if not in shadow.
  
          cudaVector3D o = hit_p + EPS_D * dir_to_light;
          float dist = dist_to_light - EPS_D;
  
          cudaRay shadow = cudaRay(o, dir_to_light, dist, 0);
          shadow.min_t = EPS_D;
  
         // if(!pathtracer->bvh->intersect(shadow))
         if(!cudaintersectWithNode(shadow, &isect, primitives, root)) 
           L_out += 1.0*(cos_theta / (num_light_samples * pr)) * f * light_L;
        }
      }
  //   }
  
  
    return L_out;
  
  }

  __device__ cudaRay generate_ray_cuda(cudaCamera* camera, double x, double y) {
    // TODO (PathTracer):
    // compute position of the input sensor sample coordinate on the
    // canonical sensor plane one unit away from the pinhole.
    x -= 0.5;
    y -= 0.5;
  //	printf("screen:%f %f %f\n", vFov, hFov, ar);

    cudaVector3D vec = cudaVector3D(x * sensorWidth, y * sensorHeight, -1);
    return cudaRay(camera->pos, camera->c2w * vec.unit());
  }

  __global__ void raytrace_pixel(cudaCamera* camera, cudaSpectrum* spectrum_buffer, cudaTriangle* primitives, 
    cudaComplexLight* lights, hiprandState *state, cudaBVHNode* root) 
  {
    // Sample the pixel with coordinate (x,y) and return the result spectrum.
    // The sample rate is given by the number of camera rays per pixel.

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t x = index % width;
    size_t y = index / width;
//    printf("index:%d ", index);
    double px, py;

    px = (x + 0.5) / width;
    py = (y + 0.5) / height;
    
//    double color = (double)index / (width*height); 
//     printf("color%g\n",color);
    // if(x < width && y < height)
    // {
    //   spectrum_buffer[y*width+x].r = color;
    //   spectrum_buffer[y*width+x].g = color;
    //   spectrum_buffer[y*width+x].b = color;
    // }
   if(x < width && y < height)
      spectrum_buffer[y * width + x] = trace_ray(generate_ray_cuda(camera, px, py), primitives, lights, state, index, root);
    //   return trace_ray(pathtracer->camera->generate_ray(px, py));

  }

__global__ void setup_random_kernel(hiprandState *state,unsigned long seed)
{
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;

  hiprand_init(seed, index, 0, &state[index]);
}
  
void cudaPathTracer::start_raytracing() {
 //   pathtracer->start_raytracing();
 hipError_t err;
    pathtracer->rayLog.clear();
    pathtracer->workQueue.clear();
  
    pathtracer->state = pathtracer->RENDERING;
    pathtracer->continueRaytracing = true;
    pathtracer->workerDoneCount = 0;
   
    pathtracer->sampleBuffer.clear();
    pathtracer->frameBuffer.clear();
    pathtracer->num_tiles_w = pathtracer->sampleBuffer.w / pathtracer->imageTileSize + 1;
    pathtracer->num_tiles_h = pathtracer->sampleBuffer.h / pathtracer->imageTileSize + 1;
    pathtracer->tile_samples.resize(pathtracer->num_tiles_w * pathtracer->num_tiles_h);
    memset(&pathtracer->tile_samples[0], 0, pathtracer->num_tiles_w * pathtracer->num_tiles_h * sizeof(int));
  
    // launch threads
    fprintf(stdout, "[CudaPathTracer] Rendering... ");
    fflush(stdout);
        
    Timer timer;
    timer.start();
    

    size_t w = pathtracer->sampleBuffer.w;
    size_t h = pathtracer->sampleBuffer.h;
    // TODO: make cuda here
    size_t blockNum = (w * h + BLOCKSIZE - 1) / BLOCKSIZE;
    //printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!%d\n", w*h);
    Spectrum* buffer = (Spectrum*)malloc(w*h * sizeof(cudaSpectrum));

    memset(buffer, 0,w*h * sizeof(cudaSpectrum));

    // TODO : D E B U G !!!!!!

    err = hipPeekAtLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to  UNKOWN (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    /* Random Number Generator */
    hiprandState *state;
    hipMalloc((void **)&state, w*h*sizeof(hiprandState));
    setup_random_kernel<<<blockNum,BLOCKSIZE>>>(state, unsigned(time(NULL)));
    raytrace_pixel<<<blockNum, BLOCKSIZE>>>(camera, spectrum_buffer, primitives, cudaLights, state, root); 
    hipDeviceSynchronize();

    err = hipPeekAtLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel raytrace_pixel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
      printf("launch success\n");

  //    err = hipMemcpy(spectrum_buffer, buffer, w*h* sizeof(cudaSpectrum), hipMemcpyHostToDevice);
    err = hipMemcpy(buffer, spectrum_buffer, w*h* sizeof(cudaSpectrum), hipMemcpyDeviceToHost);
    

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy spectrum (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for (size_t y = 0; y < h; y ++) {
      for (size_t x = 0; x < w; x ++) {
         //   spectrum_buffer[y*w+x] = raytrace_pixel(x, y);
             pathtracer->sampleBuffer.update_pixel(buffer[y*w+x], x, y);  
            // Spectrum s = raytrace_pixel(x, y);
            // pathtracer->sampleBuffer.update_pixel(s, x, y);     
       }  
    }

    pathtracer->sampleBuffer.toColor(pathtracer->frameBuffer, 0, 0, pathtracer->sampleBuffer.w, pathtracer->sampleBuffer.h);
    timer.stop();
    fprintf(stdout, "Done! (%.4fs)\n", timer.duration());
    pathtracer->state = pathtracer->DONE;

    free(buffer);

  }

  void cudaPathTracer::save_image(string fname) {
      pathtracer->save_image(fname);
    // if (state != DONE) return;
  
    // uint32_t *frame = &frameBuffer.data[0];
    // size_t w = frameBuffer.w;
    // size_t h = frameBuffer.h;
    // uint32_t *frame_out = new uint32_t[w * h];
    // for (size_t i = 0; i < h; ++i) {
    //   memcpy(frame_out + i * w, frame + (h - i - 1) * w, 4 * w);
    // }
  
    // fprintf(stderr, "[PathTracer] Saving to file: %s... ", fname.c_str());
    // lodepng::encode(fname, (unsigned char *)frame_out, w, h);
    // fprintf(stderr, "Done!\n");
  }

  bool cudaPathTracer::is_done() {
    pathtracer->update_screen();
    return pathtracer->state == pathtracer->DONE;
}
 
  bool cudaPathTracer::is_done_headless() {
        return pathtracer->state == pathtracer->DONE;

  }
