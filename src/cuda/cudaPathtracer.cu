#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CMU462/CMU462.h"
#include "CMU462/vector3D.h"
#include "CMU462/matrix3x3.h"
#include "CMU462/lodepng.h"

#include "../static_scene/sphere.h"
#include "../static_scene/triangle.h"
#include "../static_scene/light.h" 

#include "cudaPathtracer.h"


#include "cudaSpectrum.h"
#include "cudabsdf.h"
#include "cudaintersection.h"
#include "cudaPrimitive.h" 
#include "cudaTriangle.h"
#include "cudaCamera.h"
#include "cudaMatrix3x3.h"
#include "cudaRay.h"



using namespace CMU462;
using namespace StaticScene;

using std::min;
using std::max;

#define BLOCKSIZE 256
cudaTriangle* primitives;
__constant__ int primitiveCount;
__constant__ double sensorHeight; 
__constant__ double sensorWidth; 
__constant__ size_t width;
__constant__ size_t height;

PathTracer* pathtracer;
cudaSpectrum* spectrum_buffer;
cudaPrimitive* cudaPrimitives;
cudaCamera* camera;  
cudaMatrix3x3 c2w;


cudaPathTracer::cudaPathTracer(PathTracer* _pathTracer) {
    pathtracer = _pathTracer;
 
}

cudaPathTracer::~cudaPathTracer() {
    hipFree(spectrum_buffer);
    hipFree(camera);
    // delete bvh;
    // delete gridSampler; 
    // delete hemisphereSampler;
}

void loadPrimitives()
{
  hipError_t err;
  int prim_num = pathtracer->bvh->primitives.size(); 
  hipMemcpyToSymbol(HIP_SYMBOL(primitiveCount), &prim_num,  sizeof(int));
  err = hipPeekAtLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to init primitive count (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }


  hipMalloc(&primitives, sizeof(cudaTriangle) * prim_num);
  err = hipPeekAtLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to malloc primitive (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  cudaTriangle* cpuTriangle = (cudaTriangle *)malloc(prim_num * sizeof(cudaTriangle));
  // TODO: NEED TO REALLY TRANSLATE IT
  for(int i = 0; i < prim_num; i++)
  {
    Triangle* prim = (Triangle*)pathtracer->bvh->primitives[i];
    cpuTriangle[i].mesh = prim->mesh;
    cpuTriangle[i].v1 = prim->v1;
    cpuTriangle[i].v2 = prim->v2;
    cpuTriangle[i].v3 = prim->v3;
    cpuTriangle[i].v = prim->v;

    
    cpuTriangle[i].p0 = prim->mesh->positions[prim->v1]; 
    cpuTriangle[i].p1 = prim->mesh->positions[prim->v2];
    cpuTriangle[i].p2 = prim->mesh->positions[prim->v3];

    cpuTriangle[i].n0 = prim->mesh->normals[prim->v1]; 
    cpuTriangle[i].n1 = prim->mesh->normals[prim->v2]; 
    cpuTriangle[i].n2 = prim->mesh->normals[prim->v3]; 

  //  printf("%f \n",cpuTriangle[i].mesh->positions[cpuTriangle[i].v1].x);
  }
  hipMemcpy(primitives, cpuTriangle, sizeof(cudaTriangle)  * prim_num, hipMemcpyHostToDevice);
//  hipMemcpyToSymbol(HIP_SYMBOL(primitives), cpuTriangle,  sizeof(cudaTriangle)  * prim_num);
  free(cpuTriangle);

  err = hipPeekAtLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to init primitive (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
}

void cudaPathTracer::set_scene(Scene *scene) {
  hipError_t err;
  double sh = 2 * tan(radians(pathtracer->camera->vFov) / 2) * 1;	// distance is always 1
  double sw = sh * pathtracer->camera->ar;


  hipMemcpyToSymbol(HIP_SYMBOL(sensorHeight), &sh,  sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(sensorWidth), &sw,  sizeof(double));


  int num = pathtracer->sampleBuffer.w * pathtracer->sampleBuffer.h;;
  //  spectrum_buffer = (Spectrum*)malloc(sizeof(Spectrum) * num);

    hipMalloc(&spectrum_buffer, sizeof(cudaSpectrum) * num);


    hipMalloc(&camera, sizeof(cudaCamera));
    hipMemcpy(camera, pathtracer->camera, sizeof(cudaCamera), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(width), &pathtracer->sampleBuffer.w,  sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(height), &pathtracer->sampleBuffer.h,  sizeof(double));
    
    err = hipPeekAtLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to init scene (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    loadPrimitives();
 // hipMalloc(&cudaPrimitives, sizeof(cudaPrimitive) * prim_num);





  //pathtracer->set_scene(scene);
    // if (state != INIT) {
    // return;
    // }

    // if (this->scene != nullptr) {
    // delete scene;
    // delete bvh;
    // selectionHistory.pop();
    // }

    // if (this->envLight != nullptr) {
    // scene->lights.push_back(this->envLight);
    // }

    // this->scene = scene;
    // build_accel();

    // if (has_valid_configuration()) {
    // state = READY;
    // }
}

void cudaPathTracer::set_camera(Camera *camera) {
  //  pathtracer->set_camera(camera);
    // if (state != INIT) {
    // return;
    // }

    // this->camera = camera;
    // if (has_valid_configuration()) {
    // state = READY;
    // }
}

void cudaPathTracer::set_frame_size(size_t width, size_t height) {
    pathtracer->set_frame_size(width, height); 
    // if (state != INIT && state != READY) {
    // stop();
    // }
    // sampleBuffer.resize(width, height);
    // frameBuffer.resize(width, height);
    // if (has_valid_configuration()) {
    // state = READY;
    // }
}


void cudaPathTracer::update_screen() {
    pathtracer->update_screen(); 
    // switch (state) {
    //   case INIT:
    //   case READY:
    //     break;
    //   case VISUALIZE:
    //     visualize_accel();
    //     break;
    //   case RENDERING:
    //     glDrawPixels(frameBuffer.w, frameBuffer.h, GL_RGBA, GL_UNSIGNED_BYTE,
    //                  &frameBuffer.data[0]);
    //     break;
    //   case DONE:
    //     // sampleBuffer.tonemap(frameBuffer, tm_gamma, tm_level, tm_key, tm_wht);
    //     glDrawPixels(frameBuffer.w, frameBuffer.h, GL_RGBA, GL_UNSIGNED_BYTE,
    //                  &frameBuffer.data[0]);
    //     break;
    // }
  }


__device__ bool cudaintersectPrimitive(cudaTriangle* primitive, const cudaRay &r, cudaIntersection *isect)
{
  
  // size_t v1 = primitive->v1;
  // size_t v2 = primitive->v2;
  // size_t v3 = primitive->v3;

  //TODO MAKE VECTOR DIRECTLY IN CUDA TRIANGLE

    cudaVector3D p0 = primitive->p0; 
    cudaVector3D p1 = primitive->p1;
    cudaVector3D p2 = primitive->p2;
  
  
    cudaVector3D o = r.o;
    cudaVector3D d = r.d;
  
    // Vector3D o = Vector3D(co.x, co.y, co.z);
    // Vector3D d = Vector3D(cd.x, cd.y, cd.z);
  
    cudaVector3D e1 = p1 - p0;
    cudaVector3D e2 = p2 - p0;
    cudaVector3D s = o - p0;
  
    double denominator = dot(cross(e1, d), e2);
    if (denominator == 0)
      return false;
  
    cudaVector3D numerator = cudaVector3D(-dot(cross(s, e2), d), dot(cross(e1, d), s), -dot(cross(s, e2), e1));
    cudaVector3D ans = numerator / denominator;
  //	return true;
    // in triangle
    if (ans.x < 0 || ans.x > 1 || ans.y < 0 || ans.y > 1 ||
      1 - ans.x - ans.y < 0 || 1 - ans.x - ans.y > 1 ||
      ans.z < r.min_t || ans.z > r.max_t)
      return false;
  
    double u = ans.x;
    double v = ans.y;
    double t = ans.z;
   
  //  cudaVector3D tt = cudaVector3D(t.x, t.y, t.z); 
    r.max_t = t;
  
    isect->t = t;

    cudaVector3D n0 = primitive->n0;
    cudaVector3D n1 = primitive->n1;
    cudaVector3D n2 = primitive->n2;
 
    cudaVector3D tmp = (1 - u - v) * n0 + u * n1 + v * n2;
     isect->n = cudaVector3D(tmp.x, tmp.y, tmp.z);
     if (dot(isect->n, r.d) > 0)
      isect->n *= -1;
    isect->primitive = primitive;
  //  isect->bsdf = primitive->mesh->get_bsdf();		
     
  return true; 
}

__device__ bool cudaintersectWithNode(const cudaRay &ray, cudaIntersection *isect, cudaTriangle* primitives)
{
//	BVHNode* node = pathtracer->bvh->root;
  bool hit = false;

//  for (size_t p = 0; p < node->range; ++p) {
  for (size_t p = 0; p < primitiveCount; ++p) {
    if (cudaintersectPrimitive(&primitives[p], ray, isect))
//	if (pathtracer->bvh->primitives[node->start + p]->intersect(ray, isect))
  {
    hit = true;
  }
}

// stack<BVHNode*> s;
// 	double lt0, lt1, rt0, rt1;

// 	// TODO!!!
// //	int threadCount = 10;
// 	int pid = 0;
// 	int M[10];

// 	BVHNode* near;
// 	BVHNode* far;
	
// 	while(true)
// 	{
// 		// when it's leaf, intersect directly

// 		if(node->isLeaf())
// 		{	

// 			for (size_t p = 0; p < node->range; ++p) {
//         	if (cudaintersectPrimitive(pathtracer->bvh->primitives[node->start + p], ray, isect))
// 			//	if (pathtracer->bvh->primitives[node->start + p]->intersect(ray, isect))
// 				{
// 					hit = true;
// 				}
// 			}
// 			if(s.empty())
// 				break;
// 			node = s.top();
// 			s.pop();	
// 		}
// 		else
// 		{
// 			/* Parallel read ?*/
// 			int hitleft = (bool)node->l->bb.intersect(ray, lt0, lt1);
// 			int hitright = (bool)node->r->bb.intersect(ray, rt0, rt1);

// 			/* Use parallel and barrier to init */
// 			for(int i = 0; i <= 3; i++)
// 				M[i] = 0;

// 			// TODO: barrier here
// 			M[2*hitleft + hitright] = 1;
// 			// TODO: barrier here

// 			/* Visit both children */
// 			if(M[3] || (M[1] && M[2]))
// 			{
// 		//		printf("HERE!!\n");
// 				/* Decide which to go in first */
// 				M[pid] = 2 * (hitright && (rt0 < lt0)) - 1;

// 				/* TODO: PARLLEL SUM OVER HERE */
// 				if(M[pid] < 0)
// 				{
// 					near = node->l;
// 					far = node->r;
// 				}
// 				else
// 				{
// 					near = node->r;
// 					far = node->l;
// 				}
// 				s.push(far);
// 				node = near;

// 			}
// 			else if(M[2])
// 			{
// 			//	printf("HERELEFT\n");
// 				node = node->l;
// 			}

// 			else if(M[1])
// 			{
// 			//	printf("HERERIGHT\n");
// 				node = node->r;
// 			}

// 			else
// 			{
// 				if(s.empty())
// 					break;

// 				node = s.top();
// 				s.pop();
// 			}


// 		}

// 	}

	return hit;

}





__device__ cudaSpectrum trace_ray( const cudaRay &r, cudaTriangle* primitives) {
    cudaIntersection isect;  
   
   // if (!pathtracer->bvh->intersect(r, &isect)) {
    if (!cudaintersectWithNode(r, &isect, primitives)) {
      // if(pathtracer->envLight)
      // {
      //   Spectrum light_L = pathtracer->envLight->sample_dir(r);
      //   return light_L;
      // }
      // else
        return cudaSpectrum(0, 0, 0);
    }
    return cudaSpectrum(1, 1, 1);

    //  Spectrum L_out = isect.bsdf->get_emission();  // Le
  
    // // TODO (PathTracer):
    // // Instead of initializing this value to a constant color, use the direct,
    // // indirect lighting components calculated in the code below. The starter
    // // code overwrites L_out by (.5,.5,.5) so that you can test your geometry
    // // queries before you implement path tracing.
  
    // //L_out = Spectrum(5.f, 5.f, 5.f);
    // //DirectionalLight dl = DirectionalLight(5, 100);
    
  
    // cudaVector3D hit_p = r.o + r.d * isect.t;
    // cudaVector3D hit_n = isect.n;
  
    // // make a coordinate system for a hit point
    // // with N aligned with the Z direction.
    // cudaMatrix3x3 o2w;
    // make_coord_space(o2w, isect.n);
    // cudaMatrix3x3 w2o = o2w.T();
  
    // // w_out points towards the source of the ray (e.g.,
    // // toward the camera if this is a primary ray)
    // cudaVector3D w_out = w2o * (r.o - hit_p);
    // w_out.normalize();
  
  
    // if (!isect.bsdf->is_delta()) {
    //   Vector3D dir_to_light;
    //   float dist_to_light;
    //   float pr;
  
    //   // ### Estimate direct lighting integral
      
    //   for (SceneLight* light : pathtracer->scene->lights) {
  
    //     // no need to take multiple samples from a point/directional source
    //     int num_light_samples = light->is_delta_light() ? 1 : pathtracer->ns_area_light;
      
    //     // integrate light over the hemisphere about the normal
    //     for (int i = 0; i < num_light_samples; i++) {
  
    //       // returns a vector 'dir_to_light' that is a direction from
    //       // point hit_p to the point on the light source.  It also returns
    //       // the distance from point x to this point on the light source.
    //       // (pr is the probability of randomly selecting the random
    //       // sample point on the light source -- more on this in part 2)
    //       const Spectrum& light_L = light->sample_L(hit_p, &dir_to_light, &dist_to_light, &pr);
  
    //       // convert direction into coordinate space of the surface, where
    //       // the surface normal is [0 0 1]
    //       const Vector3D& w_in = w2o * dir_to_light;
    //       if (w_in.z < 0) continue;
  
    //         // note that computing dot(n,w_in) is simple
    //       // in surface coordinates since the normal is (0,0,1)
    //       double cos_theta = w_in.z;
            
    //       // evaluate surface bsdf
    //       const Spectrum& f = isect.bsdf->f(w_out, w_in);
  
    //       // TODO (PathTracer):
    //       // (Task 4) Construct a shadow ray and compute whether the intersected surface is
    //       // in shadow. Only accumulate light if not in shadow.
  
    //       Vector3D o = hit_p + EPS_D * dir_to_light;
    //       float dist = dist_to_light - EPS_D;
  
    //       Ray shadow = Ray(o, dir_to_light, dist, 0);
    //       shadow.min_t = EPS_D;
  
    //       if(!pathtracer->bvh->intersect(shadow))
    //         L_out += 1.0*(cos_theta / (num_light_samples * pr)) * f * light_L;
    //     }
    //   }
    // }
  
  
    // return L_out;
  
  }

  __device__ cudaRay generate_ray_cuda(cudaCamera* camera, double x, double y) {
    // TODO (PathTracer):
    // compute position of the input sensor sample coordinate on the
    // canonical sensor plane one unit away from the pinhole.
    x -= 0.5;
    y -= 0.5;
  //	printf("screen:%f %f %f\n", vFov, hFov, ar);

    cudaVector3D vec = cudaVector3D(x * sensorWidth, y * sensorHeight, -1);
    return cudaRay(camera->pos, camera->c2w * vec.unit());
  }

  __global__ void raytrace_pixel(cudaCamera* camera, cudaSpectrum* spectrum_buffer, cudaTriangle* primitives) {
    // Sample the pixel with coordinate (x,y) and return the result spectrum.
    // The sample rate is given by the number of camera rays per pixel.

    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t x = index % width;
    size_t y = index / width;
    
    double px, py;

    px = (x + 0.5) / width;
    py = (y + 0.5) / height;
    
    double color = (double)index / (width*height); 
//     printf("color%g\n",color);
    // if(x < width && y < height)
    // {
    //   spectrum_buffer[y*width+x].r = color;
    //   spectrum_buffer[y*width+x].g = color;
    //   spectrum_buffer[y*width+x].b = color;
    // }
   if(x < width && y < height)
      spectrum_buffer[y * width + x] = trace_ray(generate_ray_cuda(camera, px, py), primitives);
    //   return trace_ray(pathtracer->camera->generate_ray(px, py));

  }
  
void cudaPathTracer::start_raytracing() {
 //   pathtracer->start_raytracing();
 hipError_t err;
    pathtracer->rayLog.clear();
    pathtracer->workQueue.clear();
  
    pathtracer->state = pathtracer->RENDERING;
    pathtracer->continueRaytracing = true;
    pathtracer->workerDoneCount = 0;
   
    pathtracer->sampleBuffer.clear();
    pathtracer->frameBuffer.clear();
    pathtracer->num_tiles_w = pathtracer->sampleBuffer.w / pathtracer->imageTileSize + 1;
    pathtracer->num_tiles_h = pathtracer->sampleBuffer.h / pathtracer->imageTileSize + 1;
    pathtracer->tile_samples.resize(pathtracer->num_tiles_w * pathtracer->num_tiles_h);
    memset(&pathtracer->tile_samples[0], 0, pathtracer->num_tiles_w * pathtracer->num_tiles_h * sizeof(int));
  
    // launch threads
    fprintf(stdout, "[CudaPathTracer] Rendering... ");
    fflush(stdout);
        
    Timer timer;
    timer.start();
    

    size_t w = pathtracer->sampleBuffer.w;
    size_t h = pathtracer->sampleBuffer.h;
    // TODO: make cuda here
    size_t blockNum = (w * h + BLOCKSIZE - 1) / BLOCKSIZE;
    //printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!%d\n", w*h);
    Spectrum* buffer = (Spectrum*)malloc(w*h * sizeof(cudaSpectrum));

    memset(buffer, 0,w*h * sizeof(cudaSpectrum));

    // TODO : D E B U G !!!!!!

    err = hipPeekAtLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to  UNKOWN (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    raytrace_pixel<<<blockNum, BLOCKSIZE>>>(camera, spectrum_buffer, primitives); 
    hipDeviceSynchronize();

    err = hipPeekAtLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel raytrace_pixel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
      printf("launch success\n");

  //    err = hipMemcpy(spectrum_buffer, buffer, w*h* sizeof(cudaSpectrum), hipMemcpyHostToDevice);
    err = hipMemcpy(buffer, spectrum_buffer, w*h* sizeof(cudaSpectrum), hipMemcpyDeviceToHost);
    

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy spectrum (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for (size_t y = 0; y < h; y ++) {
      for (size_t x = 0; x < w; x ++) {
         //   spectrum_buffer[y*w+x] = raytrace_pixel(x, y);
             pathtracer->sampleBuffer.update_pixel(buffer[y*w+x], x, y);  
            // Spectrum s = raytrace_pixel(x, y);
            // pathtracer->sampleBuffer.update_pixel(s, x, y);     
       }  
    }

    pathtracer->sampleBuffer.toColor(pathtracer->frameBuffer, 0, 0, pathtracer->sampleBuffer.w, pathtracer->sampleBuffer.h);
    timer.stop();
    fprintf(stdout, "Done! (%.4fs)\n", timer.duration());
    pathtracer->state = pathtracer->DONE;

    free(buffer);

  }

  void cudaPathTracer::save_image(string fname) {
      pathtracer->save_image(fname);
    // if (state != DONE) return;
  
    // uint32_t *frame = &frameBuffer.data[0];
    // size_t w = frameBuffer.w;
    // size_t h = frameBuffer.h;
    // uint32_t *frame_out = new uint32_t[w * h];
    // for (size_t i = 0; i < h; ++i) {
    //   memcpy(frame_out + i * w, frame + (h - i - 1) * w, 4 * w);
    // }
  
    // fprintf(stderr, "[PathTracer] Saving to file: %s... ", fname.c_str());
    // lodepng::encode(fname, (unsigned char *)frame_out, w, h);
    // fprintf(stderr, "Done!\n");
  }

  bool cudaPathTracer::is_done() {
    pathtracer->update_screen();
    return pathtracer->state == pathtracer->DONE;
}
 
  bool cudaPathTracer::is_done_headless() {
        return pathtracer->state == pathtracer->DONE;

  }
