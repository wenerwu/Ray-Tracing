/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
// #include <helper_functions.h>
// #include <helper_cuda.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

__global__ void testKernel(int val)
{
    printf("[%d, %d]:\t\tValue is:%d\n",\
            blockIdx.y*gridDim.x+blockIdx.x,\
            threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
            val);
}


int testCudaPrintf() 
{

    printf("printf() is called. Output:\n\n");

    //Kernel configuration, where a two-dimensional grid and
    //three-dimensional blocks are configured.
    // dim3 dimGrid(2, 2);
    // dim3 dimBlock(2, 2, 2);
   // testKernel<<<dimGrid, dimBlock>>>(10);
   // cudaDeviceSynchronize();

    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
 //   cudaDeviceReset();

    return EXIT_SUCCESS;
}